#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <strings.h>
#include <ctype.h>
#ifdef __APPLE__
#include <OpenGL/gl.h>
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#include <GL/gl.h>
#endif
#include <hip/hip_runtime.h>

//#include <my_timer.h>
#include <aligned_allocator.h>

#include "boid.h"
#include "flock.h"
#include "vector3f.h"

static double device_time = 0.0;
double rX = 0.0;
double rY = 0.0;

__global__ void gpu_boids_kernel(int n, Flock* dev_flock) {
   //dev_flock.update();
}

void help() {
   fprintf(stderr,"./boids --help|-h --nboids|-n \n");
}

__host__ void gpu_boids(int n, Flock* h_flock) {
  // Allocate Device Memory
  Flock* dev_flock = NULL;
  hipMalloc(&dev_flock, sizeof(Boid)*n);
  // Create Cuda Events
  hipEvent_t calc1_event, calc2_event;
  hipEventCreate(&calc1_event);
  hipEventCreate(&calc2_event);
  // Copy Host Memory to Device
  hipMemcpy(dev_flock, h_flock, sizeof(Boid)*n, hipMemcpyHostToDevice);
  hipEventRecord(calc1_event);
  // Entering Kernel
  fprintf(stdout,"entering kernel.\n");
  gpu_boids_kernel<<<1,1>>>(n, dev_flock);
  hipEventRecord(calc2_event);
  // Free Device Memory
  hipFree(dev_flock);

  // Record device time
  float time;
  hipEventElapsedTime(&time, calc1_event, calc2_event);
  device_time += time;

  // Destroy event timers
  hipEventDestroy(calc1_event);
  hipEventDestroy(calc2_event);

}

void drawBoid() {
  glClearColor(0.4, 0.4, 0.4, 0.4);
  glClear(GL_COLOR_BUFFER_BIT);
  glColor3f(1.0, 1.0, 1.0);
  glOrtho(-1.0, 1.0, -1.0, 1.0, -1.0, 1.0);

  glBegin(GL_TRIANGLES);
    glVertex3f(-0.7, 0.7, 0);
    glVertex3f(0.7, 0.7, 0);
    glVertex3f(0, -1, 0);
  glEnd();

  glFlush();
  glutSwapBuffers();
}

void Render() {
  // Set Background Color
  glClearColor(0.4, 0.4, 0.4, 1.0);
  // Clear screen
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

  // Reset transformations
  glLoadIdentity();
  
  // Perspective modifications
  glRotatef(rX, 1.0, 0.0, 0.0);
  glRotatef(rY, 0.0, 1.0, 0.0);

  drawBoid();
}

void Keyboard(int key, int x, int y) {
  if (key == GLUT_KEY_RIGHT) {
    rY += 15;
  } else if (key == GLUT_KEY_LEFT) {
    rY -= 15;
  } else if (key == GLUT_KEY_DOWN) {
    rX -= 15;
  } else if (key == GLUT_KEY_UP) {
    rX += 15;
  }

  // Request display update
  glutPostRedisplay();		  
}

void GLInit(int argc, char* argv[]) {
  glutInit(&argc, argv);
  glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH );
  glutInitWindowSize(700, 700);
  glutInitWindowPosition(20, 20);
  glutCreateWindow("cuda-boids");
}

// Called when OpenGL Window is resized to handle scaling
void windowResize(int height, int width) {
  glViewport(0, 0, width, height);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluPerspective(45.0, (double)width / (double)height, 1.0, 200.0);
}

int main (int argc, char* argv[]) {
   /* Define the number of boids. The default is 1000. */
   int n = 1000;

   for (int i = 1; i < argc; ++i) {
#define check_index(i,str) \
   if ((i) >= argc) \
      { fprintf(stderr,"Missing 2nd argument for %s\n", str); return 1; }

      if ( strcmp(argv[i],"-h") == 0 || strcmp(argv[i],"--help") == 0) {
         help();
         return 1;
      }
      else if (strcmp(argv[i],"--nboids") == 0 || strcmp(argv[i],"-n") == 0) {
         check_index(i+1,"--nboids|-n");
         i++;
         if (isdigit(*argv[i]))
            n = atoi( argv[i] );
      } else {
         fprintf(stderr,"Unknown option %s\n", argv[i]);
         help();
         return 1;
      }
   }

   //  Memory Allocation
   fprintf(stdout, "Allocating memory for flock.\n");

   Flock* flock = NULL;
   Allocate(flock, sizeof(Boid)*n);

   {
      hipDeviceProp_t props;
      hipGetDeviceProperties( &props, 0 );
      fprintf(stderr, "   name:                           %s\n", props.name);
      fprintf(stderr, "   major.minor:                    %d.%d\n", props.major, props.minor);
      fprintf(stderr, "   totalGlobalMem:                 %lu (MB)\n", props.totalGlobalMem / (1024*1024));
      fprintf(stderr, "   sharedMemPerBlock:              %lu (KB)\n", props.sharedMemPerBlock / 1024);
      fprintf(stderr, "   sharedMemPerMultiprocessor:     %lu (KB)\n", props.sharedMemPerMultiprocessor / 1024);
      fprintf(stderr, "   regsPerBlock:                   %d\n", props.regsPerBlock);
      fprintf(stderr, "   warpSize:                       %d\n", props.warpSize);
      fprintf(stderr, "   multiProcessorCount:            %d\n", props.multiProcessorCount);
      fprintf(stderr, "   maxThreadsPerBlock:             %d\n", props.maxThreadsPerBlock);
   }

   //double t_gpu = 0, t_host = 0;
   //myTimer_t t_start = getTimeStamp();
  
   // OpenGL / GLUT
   GLInit(argc, argv);
   glutReshapeFunc(windowResize);
   glutDisplayFunc(Render);
   glutSpecialFunc(Keyboard);

   glutMainLoop();

   gpu_boids(n, flock);
   //fprintf(stdout, device_time);  
   // Memory Deallocation
   fprintf(stdout, "De-Allocating memory.\n");
   Deallocate(flock);
   return 0;
}

