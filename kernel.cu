#include "hip/hip_runtime.h"

#include "kernel.h"

dim3 threadsPerBlock(BlockSize);

const float boidMass = 1.0f;
const float scene_scale = 4e2;
__device__ hiprandState_t state;
const __device__ float neighborRadius = 20.0f;
const __device__ float neighborAngle = 180.0f;
const __device__ float c_alignment = 2.0f;
const __device__ float c_separation = 3.0f;
const __device__ float c_cohesion = 0.005f;
const __device__ float c_seek = 0.001f;

float4* dev_pos;
float3* dev_vel;
float3* dev_acc;

__device__
float distanceFormula(float3 myPos, float3 theirPos) {
  float dx = myPos.x - theirPos.x;
  float dy = myPos.y - theirPos.y;
  float dz = myPos.z - theirPos.z;

  float dist = sqrt(dx*dx + dy*dy + dz*dz);
  return dist;
}

__device__
float dotProduct(float3 v1, float3 v2) {
  return (v1.x * v2.x) + (v1.y * v2.y) + (v1.z * v2.z);
}

__device__
void add2Vectors(float3 v1, float3 v2) {
  v1.x += v2.x;
  v1.y += v2.y;
  v1.z += v2.z;
}

__device__
float3 add2VectorsNew(float3 v1, float3 v2) {
  float3 temp = make_float3(v1.x, v1.y, v1.z);
  temp.x += v2.x;
  temp.y += v2.y;
  temp.z += v2.z;
  return temp;
}

__device__
void sub2Vectors(float3 v1, float3 v2) {
  v1.x -= v2.x;
  v1.y -= v2.y;
  v1.z -= v2.z;
}

__device__
float3 sub2VectorsNew(float3 v1, float3 v2) {
  float3 temp = make_float3(v1.x, v1.y, v1.z);
  temp.x -= v2.x;
  temp.y -= v2.y;
  temp.z -= v2.z;
  return temp;
}

__device__
void mulVectorByScalar(float scalar, float3 vector) {
  vector.x *= scalar;
  vector.y *= scalar;
  vector.z *= scalar;
}

__device__
float3 mulVectorByScalarNew(float scalar, float3 vector) {
  // Temp to not overwrite original vector
  float3 temp = make_float3(vector.x, vector.y, vector.z);
  temp.x *= scalar;
  temp.y *= scalar;
  temp.z *= scalar;
  return temp;
}

__device__
void divVectorByScalar(float scalar, float3 vector) {
  vector.x /= scalar;
  vector.y /= scalar;
  vector.z /= scalar;
}

__device__
float3 divVectorByScalarNew(float scalar, float3 vector) {
  float3 temp = make_float3(vector.x, vector.y, vector.z);
  temp.x /= scalar;
  temp.y /= scalar;
  temp.z /= scalar;
  return temp;
}

__device__
void addVectorByScalar(float scalar, float3 vector) {
  vector.x += scalar;
  vector.y += scalar;
  vector.z += scalar;
}

__device__
float magnitudeOfVector(float3 vector) {
  return sqrt(vector.x*vector.x + vector.y*vector.y + vector.z*vector.z);
}

__device__
void normalizeVector(float3 vector) {
  float magnitude = magnitudeOfVector(vector);
  if (magnitude > 0) {
    vector.x /= magnitude;
    vector.y /= magnitude;
    vector.z /= magnitude;
  }
}

__device__
float3 truncate(float3 direction, float maxLength) {
  if (magnitudeOfVector(direction) > maxLength) {
    normalizeVector(direction);
    mulVectorByScalar(maxLength, direction);
    return direction;
  } else {
    return direction;
  }
}

__global__
void generateRandomPosArray(int n, float4* arr, float mass) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index < n) {
    hiprand_init(0, blockIdx.x, 0, &state);

    arr[index].x = hiprand(&state) % 20;
    arr[index].y = hiprand(&state) % 20;
    arr[index].z = hiprand(&state) % 20;

    arr[index].x = index;
    arr[index].y = index;
    arr[index].z = index;
    arr[index].w = mass;
  }
}

__global__
void generateRandomVelArray(int n, float3* arr) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index < n) {
    hiprand_init(0, blockIdx.x, 0, &state);

    arr[index].x = hiprand(&state) % 3;
    arr[index].y = hiprand(&state) % 3;
    arr[index].z = hiprand(&state) % 3;
  }
}

__global__
void updateAccelaration(int n, float4* pos, float3* vel,
                        float3* acc, float3 target) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if( index < n ) {

    float3 myPosition = make_float3(pos[index].x, pos[index].y, pos[index].z);
    float3 myVelocity = make_float3(vel[index].x, vel[index].y, vel[index].z);

    int numberOfNeighbors = 0;
    float3 alignmentNumerator = make_float3(0.0f, 0.0f, 0.0f);
    float3 alignmentVelocity = make_float3(0.0f, 0.0f, 0.0f);
    float3 separationVel = make_float3(0.0f, 0.0f, 0.0f);
    float3 centerOfMass = make_float3(0.0f, 0.0f, 0.0f);
    float3 desiredVel = make_float3(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < n; i++) {
      float3 theirPos = make_float3(pos[i].x, pos[i].y, pos[i].z);
      float distanceToNeighbor = distanceFormula(myPosition, theirPos);

      if (distanceToNeighbor > 0 && distanceToNeighbor < neighborRadius) {
	      normalizeVector(myVelocity);
	      float3 deltaPos = sub2VectorsNew(theirPos, myPosition);
        normalizeVector(deltaPos);
      	if (dotProduct(myVelocity, deltaPos) > cos(neighborAngle / 2)) {
          add2Vectors(alignmentNumerator, vel[i]);

      	  float3 reverseDelta = sub2VectorsNew(myPosition, theirPos);
      	  divVectorByScalar(distanceToNeighbor * distanceToNeighbor, reverseDelta);
          add2Vectors(separationVel, reverseDelta);

          add2Vectors(centerOfMass, theirPos);
          numberOfNeighbors++;
      	}
      }
    }

    if (numberOfNeighbors > 0) {
      alignmentVelocity = divVectorByScalarNew(float(numberOfNeighbors), alignmentNumerator);

      divVectorByScalar(float(numberOfNeighbors), centerOfMass);


      float3 alignmentVelocityTemp = mulVectorByScalarNew(c_alignment, alignmentVelocity);

      float3 separationVelTemp = mulVectorByScalarNew(c_separation, separationVel);

      float3 deltaCenterTemp = sub2VectorsNew(centerOfMass, myPosition);
      mulVectorByScalar(c_cohesion, deltaCenterTemp);

      float3 deltaTargetPosTemp = sub2VectorsNew(target, myPosition);
      normalizeVector(deltaTargetPosTemp);
      mulVectorByScalar(c_seek, deltaTargetPosTemp);

      float3 sum = make_float3(0, 0, 0);
      add2Vectors(sum, alignmentVelocityTemp);
      add2Vectors(sum, separationVelTemp);
      add2Vectors(sum, deltaCenterTemp);
      add2Vectors(sum, deltaTargetPosTemp);

      desiredVel.x = sum.x;
      desiredVel.y = sum.y;
      desiredVel.z = sum.z;
    } else {
      float3 deltaPosTarget = sub2VectorsNew(target, myPosition);
      desiredVel = mulVectorByScalarNew(c_seek, deltaPosTarget);
    }
    if (magnitudeOfVector(myPosition) > 800.0f) {
      float3 neg_myPosition = make_float3(-myPosition.x, -myPosition.y, -myPosition.z);
      normalizeVector(neg_myPosition);
      desiredVel = neg_myPosition;
    }
    // Calc acc from steering vel
    float3 deltaVel = sub2VectorsNew(desiredVel, myVelocity);
    desiredVel = truncate(deltaVel, 2.0f);
    divVectorByScalar(pos[index].w, desiredVel);
    acc[index] = desiredVel;
  }
}

__global__
void updatePosition(int n, float dt, float4 *pos, float3 *vel, float3 *acc) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if( index < n ) {

    float3 temp = mulVectorByScalarNew(dt, acc[index]);
    float3 sumTemp = add2VectorsNew(vel[index], temp);
    normalizeVector(sumTemp);
    mulVectorByScalar(2.0f, sumTemp);

    vel[index].x += sumTemp.x;
    vel[index].y += sumTemp.y;
    vel[index].z += sumTemp.z;

    // Runge- Kutta Method for ODE is a possibility
    // Euler method
    pos[index].x += vel[index].x;
    pos[index].y += vel[index].y;
    pos[index].z += vel[index].z;
  }
}

__global__
void sendToVBO(int n, float4* pos, float3* vel,
               float* posVBO, float* velVBO, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale_w = 2.0f / s_scale;
  float c_scale_h = 2.0f / s_scale;
	float c_scale_s = 2.0f / s_scale;

  if (index < n) {
    posVBO[4 * index + 0] = pos[index].x * c_scale_w;
    posVBO[4 * index + 1] = pos[index].y * c_scale_h;
    posVBO[4 * index + 2] = pos[index].z * c_scale_s;
    posVBO[4 * index + 3] = 1;

    velVBO[3 * index + 0] = vel[index].x * c_scale_w;
		velVBO[3 * index + 1] = vel[index].y * c_scale_h;
		velVBO[3 * index + 2] = vel[index].z * c_scale_s;
  }
}

__host__
void initCuda(int n) {
  //fprintf(stdout, "   Initializing CUDA.\n");
  dim3 fullBlocksPerGrid((int)ceil(float(n)/float(BlockSize)));

  checkCudaErrors( hipMalloc((void**)&dev_pos, n*sizeof(float4)) );
  checkCudaErrors( hipMalloc((void**)&dev_vel, n*sizeof(float3)) );
  checkCudaErrors( hipMalloc((void**)&dev_acc, n*sizeof(float3)) );

  generateRandomPosArray<<<fullBlocksPerGrid, BlockSize>>>(n, dev_pos, boidMass);
  generateRandomVelArray<<<fullBlocksPerGrid, BlockSize>>>(n, dev_vel);
}

__host__
void cudaFlockingUpdateWrapper(int n, float dt, float3 target) {
  dim3 fullBlocksPerGrid((int)ceil(float(n)/float(BlockSize)));

  //fprintf(stdout, "   Updating Acceleration and position.\n");
  updateAccelaration<<<fullBlocksPerGrid, BlockSize>>>(n, dev_pos, dev_vel, dev_acc, target);
  updatePosition<<<fullBlocksPerGrid, BlockSize>>>(n, dt, dev_pos, dev_vel, dev_acc);

}

__host__
void cudaUpdateVBO(int n, float* vbodptr, float* velptr) {
  dim3 fullBlocksPerGrid((int)ceil(float(n)/float(BlockSize)));

  //fprintf(stdout, "   Sending changes to VBO.\n");
  sendToVBO<<<fullBlocksPerGrid, BlockSize>>>(n, dev_pos, dev_vel, vbodptr, velptr, scene_scale);
}
